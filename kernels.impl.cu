#include "hip/hip_runtime.h"
#define SUFFIXED2(real, name) name ## _ ## real
#define SUFFIXED1(real, name) SUFFIXED2(real, name)
#define SUFFIXED(name) SUFFIXED1(real, name)

extern "C" __global__ void SUFFIXED(blend)(
        const size_t m, const size_t n, const size_t ld, const real w,
        sloped<real> *uh, sloped<real> *uhu, sloped<real> *uhv,
        const sloped<real> *oh, const sloped<real> *ohu, const sloped<real> *ohv
    )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    size_t xy = x + ld * y;
    real wu = 1 - w;
    if (x < m && y < n) {
        uh [xy] = wu * uh [xy] + w * oh [xy];
        uhu[xy] = wu * uhu[xy] + w * ohu[xy];
        uhv[xy] = wu * uhv[xy] + w * ohv[xy];
    }
}
